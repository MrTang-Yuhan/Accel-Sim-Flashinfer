#include <hip/hip_runtime.h>
#include <iostream>

#define WARPSIZE 32
// 内核函数，使用 __ballot_sync() 和 __activemask()
__global__ void kernel_example(int *g_results) {
    // 获取 warp 中的线程 ID
    int laneId = threadIdx.x % WARPSIZE;

    // 获取 warp 中活跃线程的掩码
    unsigned activeMask = __activemask();
    printf("laneId=%d, active_mask=%d\n", laneId, activeMask);

    // 每个线程设置一个值，如果其线程 ID 是偶数
    int value = (laneId % 2 == 0) ? 1 : 0;

    // 使用 __ballot_sync() 收集 warp 中所有线程的值
    int ballot = __ballot_sync(activeMask, value);

    // 计算 warp 中值为 1 的线程数量
    int count = __popc(ballot);

    // 只有 warp 中第一个线程会写结果到全局内存
    if (laneId == 0) {
        // 计算全局索引
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // 保存结果
        g_results[idx] = count;
    }
}

// 主函数
int main() {
    // 定义和分配全局内存数组来存储结果
    int *g_results = nullptr;
    size_t size = 256; // 假设我们有 256 个线程
    hipMalloc(&g_results, size * sizeof(int));

    // 定义每个块的线程数和块数
    int threadsPerBlock = 32;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // 调用内核函数
    kernel_example<<<blocksPerGrid, threadsPerBlock>>>(g_results);

    // 定义和分配主机内存数组来存储结果
    int *h_results = new int[size];

    // 从设备复制结果到主机
    hipMemcpy(h_results, g_results, size * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    // for (int i = 0; i < size; ++i) {
    //     std::cout << "Result[" << i << "] = " << h_results[i] << std::endl;
    // }

    // 释放内存
    hipFree(g_results);
    delete[] h_results;

    return 0;
}